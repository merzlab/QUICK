#include "hip/hip_runtime.h"
#include "gpu_utils.h"

#include <stdio.h>
#include <assert.h>
#if defined(MPIV_GPU)
  #include <mpi.h>
#endif


/* Safe wrapper around hipGetDeviceCount
 *
 * count: num. of GPUs on system
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuGetDeviceCount(int * count, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipGetDeviceCount(count);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipGetDeviceCount failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }  
}


/* Safe wrapper around hipSetDevice
 *
 * device: ID to device to set for execution
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuGetDeviceCount(int device, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipSetDevice(device);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        if (ret == hipErrorInvalidDevice) {
            fprintf(stderr, "[ERROR] invalid GPU device ID set (%d).\n", device);
        } else if (ret == hipErrorContextAlreadyInUse) {
            fprintf(stderr, "[ERROR] GPU device with specified ID already in use (%d).\n", device);
        }

        fprintf(stderr, "[ERROR] GPU error: hipSetDevice failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }  
}


/* Safe wrapper around hipMalloc
 *
 * ptr: pointer to allocated device memory
 * size: reqested allocation size in bytes
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMalloc(void **ptr, size_t size, const char * const filename,
        int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    fprintf(stderr, "[INFO] gpuMalloc: requesting %zu bytes at line %d in file %.*s on MPI processor %d\n",
            size, line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuMalloc: requesting %zu bytes at line %d in file %.*s\n",
            size, line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif

    ret = hipMalloc(ptr, size);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMalloc failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }  

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    fprintf(stderr, "[INFO] gpuMalloc: granted memory at address %p at line %d in file %.*s on MPI processor %d\n",
            *ptr, line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuMalloc: granted memory at address %p at line %d in file %.*s\n",
            *ptr, line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif
}


/* Safe wrapper around hipHostAlloc
 *
 * ptr: pointer to allocated device memory
 * size: reqested allocation size in bytes
 * flags: requested properties of allocated memory
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuHostAlloc(void **ptr, size_t size, unsigned int flags, const char * const filename,
        int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    fprintf(stderr, "[INFO] gpuHostAlloc: requesting %zu bytes at line %d in file %.*s on MPI processor %d\n",
            size, line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuHostAlloc: requesting %zu bytes at line %d in file %.*s\n",
            size, line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif

    ret = hipHostAlloc(ptr, size, flags);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipHostAlloc failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }  

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    fprintf(stderr, "[INFO] gpuHostAlloc: granted memory at address %p with flags %u at line %d in file %.*s on MPI processor %d\n",
            *ptr, flags, line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuHostAlloc: granted memory at address %p with flags %u at line %d in file %.*s\n",
            *ptr, flags, line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif
}


/* Safe wrapper around hipFree
 *
 * ptr: device pointer to memory to free
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuFree(void *ptr, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    if (ptr == NULL)
    {
        fprintf(stderr, "[WARNING] trying to free the already NULL pointer\n");
        fprintf(stderr, "    [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
        return;
    }  

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    fprintf(stderr, "[INFO] gpuFree: freeing ptr at line %d in file %.*s on MPI processor %d\n",
            line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuFree: freeing ptr at line %d in file %.*s\n",
            line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif

    ret = hipFree(ptr);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[WARNING] GPU error: hipFree failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);
        fprintf(stderr, "  [INFO] Memory address: %ld\n", 
                (long int) ptr);

        return;
    }  
}


/* Safe wrapper around hipHostFree
 *
 * ptr: device pointer to memory to free
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuFreeHost(void * ptr, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    if (ptr == NULL)
    {
        fprintf(stderr, "[WARNING] trying to free the already NULL pointer\n");
        fprintf(stderr, "    [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
        return;
    }  

#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    fprintf(stderr, "[INFO] gpuFreeHost: freeing ptr at line %d in file %.*s on MPI processor %d\n",
            line, (int) strlen(filename), filename, rank);
  #else
    fprintf(stderr, "[INFO] gpuFreeHost: freeing ptr at line %d in file %.*s\n",
            line, (int) strlen(filename), filename);
  #endif
    fflush(stderr);
#endif

    ret = hipHostFree(ptr);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[WARNING] GPU error: hipHostFree failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);
        fprintf(stderr, "  [INFO] Memory address: %ld\n", 
                (long int) ptr);

        return;
    }  
}


/* Safe wrapper around hipMemset
 *
 * ptr: address to device memory for which to set memory
 * data: value to set each byte of memory
 * count: num. bytes of memory to set beginning at specified address
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMemset(void *ptr, int data, size_t count,
        const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipMemset(ptr, data, count);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMemset failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipMemsetAsync
 *
 * ptr: address to device memory for which to set memory
 * data: value to set each byte of memory
 * count: num. bytes of memory to set beginning at specified address
 * s: GPU stream to perform memset in
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMemsetAsync(void *ptr, int data, size_t count,
        hipStream_t s, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipMemsetAsync(ptr, data, count, s);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMemsetAsync failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Checks if the amount of space currently allocated to ptr is sufficient,
 * and, if not, frees any space allocated to ptr before allocating the
 * requested amount of space
 *
 * ptr: pointer to allocated device memory (if required)
 * cur_size: current allocation size in bytes
 * new_size: reqested new allocation size in bytes
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuCheckMalloc(void **ptr, size_t *cur_size, size_t new_size,
        const char * const filename, int line)
{
    assert(new_size > 0 || *cur_size > 0);

    if (new_size > *cur_size)
    {
#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
        int rank;
    
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
        fprintf(stderr, "[INFO] gpuCheckMalloc: requesting %zu bytes (%zu currently allocated) at line %d in file %.*s on MPI processor %d\n",
                new_size, *cur_size, line, (int) strlen(filename), filename, rank);
  #else
        fprintf(stderr, "[INFO] gpuCheckMalloc: requesting %zu bytes (%zu currently allocated) at line %d in file %.*s\n",
                new_size, *cur_size, line, (int) strlen(filename), filename);
  #endif
        fflush(stderr);
#endif

        if (*cur_size != 0)
        {
            _gpuFree(*ptr, filename, line);
        }

        //TODO: look into using aligned alloc's
        /* intentionally over-allocate by 20% to reduce the number of allocation operations,
         * and record the new allocation size */
        *cur_size = (size_t) ceil(new_size * 1.2);
        _gpuMalloc(ptr, *cur_size, filename, line);
    }
}


/* Safe wrapper around hipMemcpy
 *
 * dest: address to be copied to
 * src: address to be copied from
 * count: num. bytes to copy
 * dir: GPU enum specifying address types for dest and src
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMemcpy(void * const dest, void const * const src, size_t count,
        hipMemcpyKind dir, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipMemcpy(dest, src, count, dir);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMemcpy failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipMemcpyAsync
 *
 * dest: address to be copied to
 * src: address to be copied from
 * count: num. bytes to copy
 * dir: GPU enum specifying address types for dest and src
 * s: GPU stream to perform the copy in
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMemcpyAsync(void * const dest, void const * const src, size_t count,
        hipMemcpyKind dir, hipStream_t s, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipMemcpyAsync(dest, src, count, dir, s);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMemcpyAsync failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipMemcpyToSymbol
 *
 * symbol: device symbol address to be copied to
 * src: address to be copied from
 * count: num. bytes to copy
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuMemcpyToSymbol(void const * const symbol, void const * const src, size_t count,
        const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipMemcpyToSymbol(HIP_SYMBOL(symbol), src, count);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipMemcpyToSymbol failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipHostGetDevicePointer
 *
 * pdev: returned pointer from mapped memory
 * phost: request host pointer mapping
 * flags: flags for extensions (must be 0 for now)
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuHostGetDevicePointer(void ** pdev, void * const phost, unsigned int flags,
        const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipHostGetDevicePointer(pdev, phost, flags);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipHostGetDevicePointer failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around check first and reallocate if needed routine for pinned memory:
 * checks if the amount of space currently allocated to ptr is sufficient,
 * and, if not, frees any space allocated to ptr before allocating the
 * requested amount of space
 *
 * ptr: pointer to memory allocation
 * cur_size: num. of bytes currently allocated
 * new_size: num. of bytes to be newly allocated, if needed
 * flags: requested properties of allocated memory
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 * */
void _gpuHostAllocCheck(void **ptr, size_t *cur_size, size_t new_size,
        unsigned int flags, int over_alloc, double over_alloc_factor,
        const char * const filename, int line)
{
    assert(new_size > 0 || *cur_size > 0);

    if (new_size > *cur_size)
    {
#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
        int rank;
    
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
        fprintf(stderr, "[INFO] gpuHostAllocCheck: requesting %zu bytes (%zu currently allocated) with flags %u at line %d in file %.*s on MPI processor %d\n",
                new_size, *cur_size, flags, line, (int) strlen(filename), filename, rank);
  #else
        fprintf(stderr, "[INFO] gpuHostAllocCheck: requesting %zu bytes (%zu currently allocated) with flags %u at line %d in file %.*s\n",
                new_size, *cur_size, flags, line, (int) strlen(filename), filename);
  #endif
        fflush(stderr);
#endif

        if (*cur_size != 0)
        {
            _gpuFreeHost(*ptr, filename, line);
        }

        if (over_alloc == 1)
        {
            *cur_size = (int) ceil(new_size * over_alloc_factor);
        }
        else
        {
            *cur_size = new_size;
        }

        _gpuHostAlloc(ptr, *cur_size, flags, filename, line);
    }
}


/* Safe wrapper around check first and reallocate if needed
 * while preserving current memory contents routine for pinned memory:
 * checks if the amount of space currently allocated to ptr is sufficient,
 * and, if not, frees any space allocated to ptr before allocating the
 * requested amount of space
 *
 * ptr: pointer to memory allocation
 * cur_size: num. of bytes currently allocated
 * new_size: num. of bytes to be newly allocated, if needed
 * flags: requested properties of allocated memory
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 * */
void _gpuHostReallocCheck(void **ptr, size_t *cur_size, size_t new_size,
        unsigned int flags, int over_alloc, double over_alloc_factor,
        const char * const filename, int line)
{
    void *old_ptr;
    size_t old_ptr_size;

    assert(new_size > 0 || *cur_size > 0);

    if (new_size > *cur_size)
    {
#if defined(DEBUG_FOCUS)
  #if defined(MPIV_GPU)
        int rank;
    
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
        fprintf(stderr, "[INFO] gpuHostReallocCheck: requesting %zu bytes (%zu currently allocated) with flags %u at line %d in file %.*s on MPI processor %d\n",
                new_size, *cur_size, flags, line, (int) strlen(filename), filename, rank);
  #else
        fprintf(stderr, "[INFO] gpuHostReallocCheck: requesting %zu bytes (%zu currently allocated) with flags %u at line %d in file %.*s\n",
                new_size, *cur_size, flags, line, (int) strlen(filename), filename);
  #endif
        fflush(stderr);
#endif

        old_ptr = *ptr;
        old_ptr_size = *cur_size;
        *ptr = NULL;

        if (over_alloc == 1)
        {
            *cur_size = (int) ceil(new_size * over_alloc_factor);
        }
        else
        {
            *cur_size = new_size;
        }

        _gpuHostAlloc(ptr, *cur_size, flags, filename, line);

        if (old_ptr_size != 0)
        {
            _gpuMemcpy(*ptr, old_ptr, old_ptr_size, hipMemcpyHostToHost,
                    __FILE__, __LINE__);

            _gpuFreeHost(old_ptr, filename, line);
        }
    }
}


/* Safe wrapper around hipEventCreate
 *
 * event: created GPU event
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuEventCreate(hipEvent_t * event, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipEventCreate(event);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipEventCreate failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipEventDestroy
 *
 * event: GPU event to destroy
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuEventDestroy(hipEvent_t event, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipEventDestroy(event);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipEventDestroy failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipEventElapsedTime
 *
 * time: elapsed time between GPU events (in ms)
 * start, end: GPU events to compute elapsed time for
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuEventElapsedTime(float * time, hipEvent_t start, hipEvent_t end, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipEventElapsedTime(time, start, end);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipEventElapsedTime failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipEventRecord
 *
 * event: GPU event to record
 * stream: GPU stream in which to record event
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuEventRecord(hipEvent_t event, hipStream_t stream, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipEventRecord(event, stream);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipEventRecord failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipEventSynchronize
 *
 * event: GPU event to record
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuEventSynchronize(hipEvent_t event, const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipEventSynchronize(event);

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipEventSynchronize failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}


/* Safe wrapper around hipDeviceSynchronize
 *
 * filename: NULL-terminated source filename where function call originated
 * line: line of source file where function call originated
 */
void _gpuDeviceSynchronize(const char * const filename, int line)
{
#if defined(MPIV_GPU)
    int rank;
#endif
    hipError_t ret;

    ret = hipDeviceSynchronize();

    if (ret != hipSuccess)
    {
#if defined(MPIV_GPU)
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
        const char *str = hipGetErrorString(ret);

        fprintf(stderr, "[ERROR] GPU error: hipDeviceSynchronize failure\n");
#if defined(MPIV_GPU)
        fprintf(stderr, "  [INFO] At line %d in file %.*s on MPI processor %d\n",
                line, (int) strlen(filename), filename, rank);
#else
        fprintf(stderr, "  [INFO] At line %d in file %.*s\n",
                line, (int) strlen(filename), filename);
#endif
        fprintf(stderr, "  [INFO] Error code: %d\n", ret);
        fprintf(stderr, "  [INFO] Error message: %.*s\n", (int) strlen(str), str);

#if defined(MPIV_GPU)
        MPI_Abort(MPI_COMM_WORLD, 1);
#else
        exit(1);
#endif
    }
}
