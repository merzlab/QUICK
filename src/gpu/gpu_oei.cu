/*
   !---------------------------------------------------------------------!
   ! Written by Madu Manathunga on 06/17/2021                            !
   !                                                                     !
   ! Copyright (C) 2020-2021 Merz lab                                    !
   ! Copyright (C) 2020-2021 Götz lab                                    !
   !                                                                     !
   ! This Source Code Form is subject to the terms of the Mozilla Public !
   ! License, v. 2.0. If a copy of the MPL was not distributed with this !
   ! file, You can obtain one at http://mozilla.org/MPL/2.0/.            !
   !_____________________________________________________________________!

   !---------------------------------------------------------------------!
   ! This source file contains functions required for QUICK one electron !
   ! integral computation.                                               !
   !---------------------------------------------------------------------!
 */

#if defined(CUDA) || defined(CUDA_MPIV)
  #include "cuda/gpu.h"
#elif defined(HIP) || defined(HIP_MPIV)
  #include "hip/gpu.h"
#endif
#include "gpu_common.h"


static __constant__ gpu_simulation_type devSim;
static __constant__ int devTrans[TRANSDIM * TRANSDIM * TRANSDIM];
static __constant__ int Sumindex[10] = {0, 0, 1, 4, 10, 20, 35, 56, 84, 120};


#define STOREDIM 20
#define REG_PF
#define REG_FP
#define REG_SF
#define REG_FS
//#define USE_PARTIAL_DP
//#define USE_PARTIAL_PF
//#define USE_PARTIAL_FP

#include "gpu_oei_classes.h"
#include "gpu_oei_definitions.h"
#include "gpu_oei_assembler.h"
#include "gpu_oei.h"
#include "gpu_oei_grad_assembler.h"
#include "gpu_oei_grad.h"


/*
   upload trans array to constant memory
 */
void upload_para_to_const_oei() {
    int trans[TRANSDIM * TRANSDIM * TRANSDIM];

    LOC3(trans, 0, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   1;
    LOC3(trans, 0, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   4;
    LOC3(trans, 0, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  10;
    LOC3(trans, 0, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  20;
    LOC3(trans, 0, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  35;
    LOC3(trans, 0, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  56;
    LOC3(trans, 0, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) =  84;
    LOC3(trans, 0, 0, 7, TRANSDIM, TRANSDIM, TRANSDIM) = 120;
    LOC3(trans, 0, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   3;
    LOC3(trans, 0, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   6;
    LOC3(trans, 0, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  17;
    LOC3(trans, 0, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  32;
    LOC3(trans, 0, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  48;
    LOC3(trans, 0, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  67;
    LOC3(trans, 0, 1, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 100;
    LOC3(trans, 0, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   9;
    LOC3(trans, 0, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  16;
    LOC3(trans, 0, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  23;
    LOC3(trans, 0, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  42;
    LOC3(trans, 0, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  73;
    LOC3(trans, 0, 2, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 106;
    LOC3(trans, 0, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  19;
    LOC3(trans, 0, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  31;
    LOC3(trans, 0, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  43;
    LOC3(trans, 0, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  79;
    LOC3(trans, 0, 3, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 112;
    LOC3(trans, 0, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  34;
    LOC3(trans, 0, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  49;
    LOC3(trans, 0, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  74;
    LOC3(trans, 0, 4, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 113;
    LOC3(trans, 0, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  55;
    LOC3(trans, 0, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  68;
    LOC3(trans, 0, 5, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 107;
    LOC3(trans, 0, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  83;
    LOC3(trans, 0, 6, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 101;
    LOC3(trans, 0, 7, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 119;
    LOC3(trans, 1, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   2;
    LOC3(trans, 1, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =   7;
    LOC3(trans, 1, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  15;
    LOC3(trans, 1, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  28;
    LOC3(trans, 1, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  50;
    LOC3(trans, 1, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  69;
    LOC3(trans, 1, 0, 6, TRANSDIM, TRANSDIM, TRANSDIM) = 102;
    LOC3(trans, 1, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   5;
    LOC3(trans, 1, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  11;
    LOC3(trans, 1, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  26;
    LOC3(trans, 1, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  41;
    LOC3(trans, 1, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  59;
    LOC3(trans, 1, 1, 5, TRANSDIM, TRANSDIM, TRANSDIM) =  87;
    LOC3(trans, 1, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  13;
    LOC3(trans, 1, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  25;
    LOC3(trans, 1, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  36;
    LOC3(trans, 1, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  60;
    LOC3(trans, 1, 2, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  88;
    LOC3(trans, 1, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  30;
    LOC3(trans, 1, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  40;
    LOC3(trans, 1, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  61;
    LOC3(trans, 1, 3, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  94;
    LOC3(trans, 1, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  52;
    LOC3(trans, 1, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  58;
    LOC3(trans, 1, 4, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  89;
    LOC3(trans, 1, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  71;
    LOC3(trans, 1, 5, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  86;
    LOC3(trans, 1, 6, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 104;
    LOC3(trans, 2, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =   8;
    LOC3(trans, 2, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  14;
    LOC3(trans, 2, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  22;
    LOC3(trans, 2, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  44;
    LOC3(trans, 2, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  75;
    LOC3(trans, 2, 0, 5, TRANSDIM, TRANSDIM, TRANSDIM) = 108;
    LOC3(trans, 2, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  12;
    LOC3(trans, 2, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  24;
    LOC3(trans, 2, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  37;
    LOC3(trans, 2, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  62;
    LOC3(trans, 2, 1, 4, TRANSDIM, TRANSDIM, TRANSDIM) =  90;
    LOC3(trans, 2, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  21;
    LOC3(trans, 2, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  38;
    LOC3(trans, 2, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  66;
    LOC3(trans, 2, 2, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  99;
    LOC3(trans, 2, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  46;
    LOC3(trans, 2, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  64;
    LOC3(trans, 2, 3, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  98;
    LOC3(trans, 2, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  77;
    LOC3(trans, 2, 4, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  92;
    LOC3(trans, 2, 5, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 110;
    LOC3(trans, 3, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  18;
    LOC3(trans, 3, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  27;
    LOC3(trans, 3, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  45;
    LOC3(trans, 3, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  80;
    LOC3(trans, 3, 0, 4, TRANSDIM, TRANSDIM, TRANSDIM) = 114;
    LOC3(trans, 3, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  29;
    LOC3(trans, 3, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  39;
    LOC3(trans, 3, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  63;
    LOC3(trans, 3, 1, 3, TRANSDIM, TRANSDIM, TRANSDIM) =  95;
    LOC3(trans, 3, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  47;
    LOC3(trans, 3, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  65;
    LOC3(trans, 3, 2, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  97;
    LOC3(trans, 3, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  81;
    LOC3(trans, 3, 3, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  96;
    LOC3(trans, 3, 4, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 116;
    LOC3(trans, 4, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  33;
    LOC3(trans, 4, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  51;
    LOC3(trans, 4, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  76;
    LOC3(trans, 4, 0, 3, TRANSDIM, TRANSDIM, TRANSDIM) = 115;
    LOC3(trans, 4, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  53;
    LOC3(trans, 4, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  57;
    LOC3(trans, 4, 1, 2, TRANSDIM, TRANSDIM, TRANSDIM) =  91;
    LOC3(trans, 4, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  78;
    LOC3(trans, 4, 2, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  93;
    LOC3(trans, 4, 3, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 117;
    LOC3(trans, 5, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  54;
    LOC3(trans, 5, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  70;
    LOC3(trans, 5, 0, 2, TRANSDIM, TRANSDIM, TRANSDIM) = 109;
    LOC3(trans, 5, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  72;
    LOC3(trans, 5, 1, 1, TRANSDIM, TRANSDIM, TRANSDIM) =  85;
    LOC3(trans, 5, 2, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 111;
    LOC3(trans, 6, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) =  82;
    LOC3(trans, 6, 0, 1, TRANSDIM, TRANSDIM, TRANSDIM) = 103;
    LOC3(trans, 6, 1, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 105;
    LOC3(trans, 7, 0, 0, TRANSDIM, TRANSDIM, TRANSDIM) = 118;

    gpuMemcpyToSymbol((const void *) devTrans, (const void *) trans, sizeof(int) * TRANSDIM * TRANSDIM * TRANSDIM);
}


/*
   upload gpu simulation type to constant memory
 */
void upload_sim_to_constant_oei(_gpu_type gpu) {
    gpuMemcpyToSymbol((const void *) &devSim, (const void *) &gpu->gpu_sim, sizeof(gpu_simulation_type));
}


#if defined(DEBUG) || defined(DEBUGTIME)
static float totTime;
#endif


// interface for kernel launching
void getOEI(_gpu_type gpu) {
    QUICK_SAFE_CALL((getOEI_kernel<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
}


void get_oei_grad(_gpu_type gpu) {
    QUICK_SAFE_CALL((get_oei_grad_kernel<<<gpu->blocks, gpu->twoEThreadsPerBlock>>>()));
}

