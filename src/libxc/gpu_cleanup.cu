
void gpu_libxc_cleanup(gpu_libxc_info* d_glinfo, gpu_libxc_in* d_glin, gpu_libxc_out* d_glout){

	gpu_libxc_info* h_glinfo;
	gpu_libxc_in* h_glin;
	gpu_libxc_out* h_glout;

	h_glinfo = (gpu_libxc_info*)malloc(sizeof(gpu_libxc_info));
	h_glin = (gpu_libxc_in*)malloc(sizeof(gpu_libxc_in));
	h_glout = (gpu_libxc_out*)malloc(sizeof(gpu_libxc_out));

	hipMemcpy(h_glinfo, d_glinfo, sizeof(gpu_libxc_info), hipMemcpyDeviceToHost);
	hipMemcpy(h_glin, d_glin, sizeof(gpu_libxc_in), hipMemcpyDeviceToHost);
	hipMemcpy(h_glout, d_glout, sizeof(gpu_libxc_out), hipMemcpyDeviceToHost);

        hipFree(d_glout);
        hipFree(h_glout->d_zk);
        hipFree(h_glout->d_vrho);
        hipFree(h_glout->d_vsigma);

        hipFree(h_glinfo->d_maple2c_params);
        hipFree(h_glinfo->d_gdm);
        hipFree(h_glinfo->d_ds);
        hipFree(h_glinfo->d_rhoLDA);
        hipFree(h_glinfo->d_std_libxc_work_params);
        hipFree(d_glinfo);

        hipFree(h_glin->d_rho);
        hipFree(h_glin->d_sigma);
        hipFree(d_glin);

        free(h_glout);
        free(h_glinfo);
        free(h_glin);

}
