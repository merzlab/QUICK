#if defined(HIP) || defined(HIP_MPIV)
  #include "../gpu/hip/gpu_utils.h"
  #define gpuMemcpyHostToDevice hipMemcpyHostToDevice
#elif defined(CUDA) || defined(CUDA_MPIV)
  #include "../gpu/cuda/gpu_utils.h"
  #define gpuMemcpyHostToDevice hipMemcpyHostToDevice
#endif


//Uploads parameters required for kernels. *p is a pointer to libxc functional, gpu_work_params is a pointer
//to host memory location containing worker paramters
void* gpu_upload_maple2c_params(const xc_func_type *p)
{
    void *d_maple_params;

#ifdef DEBUG
    printf("FILE: %s, LINE: %d, FUNCTION: %s, p->params_byte_size: %d \n",
            __FILE__, __LINE__, __func__, p->params_byte_size);
#endif

    gpuMalloc((void **) &d_maple_params, p->params_byte_size);
    gpuMemcpy(d_maple_params, p->params, p->params_byte_size, gpuMemcpyHostToDevice);

    return d_maple_params;
}


//Uploads paramters required for kernels. *p is a pointer to libxc functional, gpu_work_params is a pointer
//to host memory location containing worker paramters
void* gpu_upload_work_params(const xc_func_type *p, void* gpu_work_params)
{
    void *d_work_params;
    int work_param_size;

    //check the family
    switch (p->info->family) {
        case XC_FAMILY_LDA:
            work_param_size = sizeof(gpu_lda_work_params);
#ifdef DEBUG
            printf("FILE: %s, LINE: %d, FUNCTION: %s, lda_work_param_size: %d \n",
                    __FILE__, __LINE__, __func__, work_param_size);
#endif
            break;

        case XC_FAMILY_HYB_GGA:
        case XC_FAMILY_GGA:
            //Now check the kind.
            switch (p->info->kind) {
                case XC_EXCHANGE:
                    work_param_size = sizeof(gpu_ggax_work_params);
                    break;

                case XC_CORRELATION:
                case XC_EXCHANGE_CORRELATION:
                    work_param_size = sizeof(gpu_ggac_work_params);
                    break;

                default:
                    break;
            }
            break;

        default:
            break;
    }

#ifdef DEBUG
    printf("FILE: %s, LINE: %d, FUNCTION: %s, gga_work_param_size: %d \n",
            __FILE__, __LINE__, __func__, work_param_size);
#endif

    gpuMalloc((void **) &d_work_params, work_param_size);
    gpuMemcpy(d_work_params, gpu_work_params, work_param_size, gpuMemcpyHostToDevice);

    return d_work_params;
}


//returns a pointer to an empty device array
double* gpu_upload_libxc_out_array(int size)
{
    double *d_double_arr;
    int arr_size = size * sizeof(double);

    gpuMalloc((void **) &d_double_arr, arr_size);

    return d_double_arr;
}


//returns a pointer to a populated device array
double* gpu_upload_libxc_input_array(const double *h_input, int size)
{
    double *d_double_arr;
    int arr_size = size * sizeof(double);

    gpuMalloc((void **) &d_double_arr, arr_size);
    gpuMemcpy(d_double_arr, h_input, arr_size, gpuMemcpyHostToDevice);

    return d_double_arr;
}


//Returns an integer that uniquly identifies the gpu worker
int get_gpu_worker(const xc_func_type *p)
{
    int gpu_wt = 0;

    //check the family
    switch (p->info->family) {
        case XC_FAMILY_LDA:
            gpu_wt = GPU_WORK_LDA;
            break;

        case XC_FAMILY_HYB_GGA:
        case XC_FAMILY_GGA:
            //Now check the kind.
            switch (p->info->kind) {
                case XC_EXCHANGE:
                    gpu_wt = GPU_WORK_GGA_X;
                    break;

                case XC_CORRELATION:
                case XC_EXCHANGE_CORRELATION:
                    gpu_wt = GPU_WORK_GGA_C;
                    break;

                default:
                    break;
            }
            break;

        default:
            break;
    }

    return gpu_wt;

}


gpu_libxc_info* gpu_upload_libxc_info(const xc_func_type *p, void *ggwp, double mix_coeff, int np)
{
    gpu_libxc_info h_glinfo;

    h_glinfo.func_id = p->info->number;
    h_glinfo.gpu_worker = get_gpu_worker(p);
    h_glinfo.mix_coeff = mix_coeff;

    h_glinfo.d_maple2c_params = gpu_upload_maple2c_params(p);
    h_glinfo.d_worker_params = gpu_upload_work_params(p, ggwp);

    //allocate device memory for some work params required by gga_x worker.
    h_glinfo.d_gdm = gpu_upload_libxc_out_array(np);
    h_glinfo.d_ds = gpu_upload_libxc_out_array(np);
    h_glinfo.d_rhoLDA = gpu_upload_libxc_out_array(np);

    gpu_libxc_info* d_glinfo;
    gpuMalloc((void **) &d_glinfo, sizeof(gpu_libxc_info));
    gpuMemcpy(d_glinfo, &h_glinfo, sizeof(gpu_libxc_info), gpuMemcpyHostToDevice);

    return d_glinfo;
}


//gpu_libxc_out* gpu_upload_libxc_out(int np)
//{
//    gpu_libxc_out h_glout;
//    h_glout.d_zk = gpu_upload_libxc_out_array(np);
//    h_glout.d_vrho = gpu_upload_libxc_out_array(np);
//    h_glout.d_vsigma = gpu_upload_libxc_out_array(np);
//
//
//    gpu_libxc_out* d_glout;
//    gpuMalloc((void **) &d_glout, sizeof(gpu_libxc_out));
//    gpuMemcpy(d_glout, &h_glout, sizeof(gpu_libxc_out), gpuMemcpyHostToDevice);
//
//    return d_glout;
//}
//
//
//gpu_libxc_in* gpu_upload_libxc_in(const double* h_rho, const double *h_sigma, int np)
//{
//    gpu_libxc_in h_glin;
//    h_glin.d_rho = gpu_upload_libxc_input_array(h_rho, np);
//    h_glin.d_sigma = gpu_upload_libxc_input_array(h_sigma, np);
//
//    gpu_libxc_in* d_glin;
//    gpuMalloc((void **) &d_glin, sizeof(gpu_libxc_in));
//    gpuMemcpy(d_glin, &h_glin, sizeof(gpu_libxc_in), gpuMemcpyHostToDevice);
//
//    return d_glin;
//}
