/*
  !---------------------------------------------------------------------!
  ! Written by Madu Manathunga on 11/08/2020                            !
  !                                                                     ! 
  ! Copyright (C) 2020-2021 Merz lab                                    !
  ! Copyright (C) 2020-2021 Götz lab                                    !
  !                                                                     !
  ! This Source Code Form is subject to the terms of the Mozilla Public !
  ! License, v. 2.0. If a copy of the MPL was not distributed with this !
  ! file, You can obtain one at http://mozilla.org/MPL/2.0/.            !
  !_____________________________________________________________________!

  !---------------------------------------------------------------------!
  ! This source file contains functions and device kernels required     !
  ! for custom matrix operations.                                       !
  !---------------------------------------------------------------------!
*/
#include "gpu.h"
#include <hip/hip_runtime.h>

static __constant__ gpu_simulation_type devSim_matop;

// upload gpu simulation type to constant memory
void upload_sim_to_constant_matop(_gpu_type gpu){
    hipError_t status;
    PRINTDEBUG("UPLOAD CONSTANT MATOP");
    status = hipMemcpyToSymbol(HIP_SYMBOL(devSim_matop), &gpu->gpu_sim, sizeof(gpu_simulation_type));
    PRINTERROR(status, " hipMemcpyToSymbol, matop sim copy to constants failed")
    PRINTDEBUG("FINISH UPLOAD CONSTANT MATOP");
}

#ifdef DEBUG
static float totTime;
#endif

void get_dmx(_gpu_type gpu){

#ifdef DEBUG
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
#endif

        QUICK_SAFE_CALL((get_dmx_kernel<<< gpu -> blocks, gpu -> threadsPerBlock>>>()));

#ifdef DEBUG
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, start, end);
    totTime+=time;
    fprintf(gpu->debugFile,"Time to form new density matrix:%f ms total time:%f ms\n", time, totTime);
    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

}


